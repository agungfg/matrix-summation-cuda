#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#define N 18

__global__ void sum(double *a, double *b, double *c) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	c[index] = a[index] + b[index];
}


void printSchema(int threadPerBlock[]) {
	int schema = 1;

	do {
		int block = 0;
		int thread = 0;

		printf("\n  Schema with %i Thread/Block", threadPerBlock[schema]);

		printf("\n |");
		for (int i = 0; i < N; i++) {
			printf("====|");
		}

		printf("\n | ");
		for (int i = 0; i < N; i++) {
			printf("%2.1d | ", i);
		}
		
		printf("\n |");
		for (int i = 0; i < N; i++) {
			printf("====|");
		}

		printf("\n | ");
		for (int i = 0; i < N; i++) {
			if (thread < threadPerBlock[schema]) {
				printf("%2.1d | ", thread);
				thread = thread + 1;
			}
			if (thread == threadPerBlock[schema]) {
				thread = 0;
				block = block + 1;
			}
		}

		printf("\n |");
		for (int i = 0; i < N; i++) {
			printf("====|");
		}

		printf("\n\n");
		schema = schema + 1;
	} while (schema <= 3);

}

int main() {
	int size = N * sizeof(double*);
	double *a = (double*)malloc(size);
	double *b = (double*)malloc(size);
	double *c = (double*)malloc(size);
	double *d_a, *d_b, *d_c;

	int schema = 3;
	int threadPerBlock[4];
	threadPerBlock[1] = 3;
	threadPerBlock[2] = 6;
	threadPerBlock[3] = 9;

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	for (int i = 0; i < N; i++) {
		a[i] = i * (3. / 4.);
		b[i] = i * (1. / 4.);
	}

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	sum << < N / threadPerBlock[schema], threadPerBlock[schema] >> > (d_a, d_b, d_c);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	printSchema(threadPerBlock);

	printf(" Results Summation \n");
	printf(" |======|=============|=============|=============| \n");
	printf(" |  No  |      A      |      B      |  C = A + B  | \n");
	printf(" |======|=============|=============|=============| \n");
	for (int i = 0; i < N; i++) {
		printf(" | %4.1d | %8.2f    | %8.2f    | %8.2f    | \n", i, a[i], b[i], c[i]);
	}
	printf(" |======|=============|=============|=============| \n");

	free(a); hipFree(d_a);
	free(b); hipFree(d_b);
	free(c); hipFree(d_c);

	getchar();

}